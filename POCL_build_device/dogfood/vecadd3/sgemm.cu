
#include <hip/hip_runtime.h>
__global__ void sgemm (float *A, float *B, float *C, int N)
{
  // Thread identifiers
  const int r = blockIdx.x; // Row ID
  const int c = blockIdx.y; // Col ID

  // Compute a single element (loop a K)
  float acc = 0.0f;
  for (int k = 0; k < N; k++) {
    acc += A[k * N + r] * B[c * N + k];
  }

  // Store the result
  C[c * N + r] = acc;
}
