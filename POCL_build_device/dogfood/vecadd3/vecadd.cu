
#include <hip/hip_runtime.h>
__global__ void vecadd(float *a, float *b, float* c) 
{
    // Get our global thread ID
    int id = blockIdx.x;
 
    // Make sure we do not go out of bounds
    c[id] = a[id] + b[id];
}
