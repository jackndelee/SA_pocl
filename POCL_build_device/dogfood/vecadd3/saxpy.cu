
#include <hip/hip_runtime.h>
__global__ void saxpy(float *src, float *dst, float factor)
{
  int i = blockIdx.x;
  dst[i] += src[i] * factor;
}

